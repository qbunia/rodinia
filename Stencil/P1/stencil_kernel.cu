#include "hip/hip_runtime.h"
#include "stencil_omp_cuda.h"
#include <stdio.h>
#define BLOCK_SIZE 16

//Each thread computes one pixel, the whole image is in global memory, filter is in global memory as well. filter size is parametered. 
//For fixed filter size, we can put the filter in either register (as passing argument of the kernel) or shared memory
__global__
void global_element(REAL* src, REAL* dst, int width, int height, const float* filter, int flt_width, int flt_height) {

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    REAL sum = 0;
    #pragma unroll
    for (int n = 0; n < flt_width; n++) {
        for (int m = 0; m < flt_height; m++) {
            int x = j + n - flt_width / 2;
            int y = i + m - flt_height / 2;
            if (x >= 0 && x < width && y >= 0 && y < height) {
                int idx = m*flt_width + n;
                sum += src[y*width + x] * filter[idx];
            }
        }
    }

    // Each thread writes one element to C matrix
    dst[i*width + j] = sum;
}

void stencil_kernel(REAL* input, REAL* output, int width, int height, const float* filter, int filter_width, int filter_height, int kernel) {
    REAL *input_device, *output_device;
    float *filter_device;
    hipMalloc(&input_device, width*height*sizeof(REAL));
    hipMalloc(&output_device, width*height*sizeof(REAL));
    hipMalloc(&filter_device, filter_width*filter_height*sizeof(float));

    hipMemcpy(input_device, input, width*height*sizeof(REAL), hipMemcpyHostToDevice);
    //hipMemcpy(output_device, output, width*height*sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(filter_device, filter, filter_width*filter_height*sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y);
    global_element<<<dimGrid, dimBlock>>>(input_device, output_device, width, height, filter_device, filter_width, filter_height);

    hipMemcpy(output, output_device, width*height*sizeof(REAL), hipMemcpyDeviceToHost);
    hipFree(input_device);
    hipFree(output_device);
    hipFree(filter_device);
}
