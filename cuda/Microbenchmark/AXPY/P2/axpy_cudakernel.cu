#include "hip/hip_runtime.h"
#include "axpy.h"

__global__ 
void
axpy_cudakernel_warmingup(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) y[i] += a*x[i];
}

/* cyclic distribution of loop distribution */
__global__
void axpy_cudakernel_P2(REAL* x, REAL* y, int n, REAL a) {
	int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
	int total_threads = gridDim.x * blockDim.x;
	
	int i;
	for (i=thread_num; i<n; i+=total_threads) { 
		if (i < n) y[i] += a*x[i];
	}
}

void axpy_cuda(REAL* x, REAL* y, int n, REAL a) {
  REAL *d_x, *d_y;
  hipMalloc(&d_x, n*sizeof(REAL));
  hipMalloc(&d_y, n*sizeof(REAL));

  hipMemcpy(d_x, x, n*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n*sizeof(REAL), hipMemcpyHostToDevice);

  // Perform axpy elements
  axpy_cudakernel_warmingup<<<(n+255)/256, 256>>>(d_x, d_y, n, a);
  hipDeviceSynchronize();
  axpy_cudakernel_P2<<<(n+255)/256, 256>>>(d_x, d_y, n, a);
  hipDeviceSynchronize();

  hipMemcpy(y, d_y, n*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
}

