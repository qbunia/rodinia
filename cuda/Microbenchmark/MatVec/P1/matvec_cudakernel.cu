#include "hip/hip_runtime.h"
#include "matvec.h"

__global__ void matvec_P1(REAL* matrix, REAL* vector, REAL* result, int n, int m)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        REAL temp = 0.0;
        for (int j = 0; j < m; j++)
            temp += matrix[i * m + j] * vector[j];
        result[i] = temp;
    }
}

void matvec_cuda(REAL* result, REAL* vector, REAL* matrix, int n, int m) {
  REAL *d_matrix, *d_vector, *d_result;
  hipMalloc(&d_matrix, n*m*sizeof(REAL));
  hipMalloc(&d_vector, m*sizeof(REAL));
  hipMalloc(&d_result, n*sizeof(REAL));

  hipMemcpy(d_matrix, matrix, n*m*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_vector, vector, m*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_result, result, n*sizeof(REAL), hipMemcpyHostToDevice);

  // Perform matvec elements
  matvec_P1<<<(n+255)/256, 256>>>(d_matrix, d_vector, d_result, n, m);

  hipMemcpy(result, d_result, n*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_matrix);
  hipFree(d_vector);
  hipFree(d_result);
}
