#include "hip/hip_runtime.h"
/* A simple lib to facilitate CUDA coding 

Extensions to the xomp interface

Liao 4/11/2012
*/

//#include "cuda_lib.h"
#include "libxomp.h"
//----------------------------------------------------
// Device xomp_cuda_property retrieving functions

#ifdef __cplusplus
extern "C" {
#endif
DDE** DDE_head;
DDE** DDE_tail;

int xomp_num_devices = -1; 
int xomp_max_num_devices =  -1; // -1 means un-initialized

void** xomp_cuda_prop; 
#ifdef __cplusplus
}
#endif

bool xomp_verbose = false;

/* Set the device id to be used by the current task */
void xomp_set_default_device (int devID)
{
  hipError_t err;
  assert (devID>=0 && devID< xomp_get_max_devices());
  err = hipSetDevice(devID); 
  if(err != hipSuccess)
  {
    fprintf(stderr,"XOMP acc_init: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(err);
  }

}

/* Obtain the max number of devices supported by the hardware*/
int xomp_get_max_devices(void)
{
  int rt; 
  hipError_t err;
  if (xomp_max_num_devices != -1)
    return xomp_max_num_devices;

  err = hipGetDeviceCount(&rt);
  if(err != hipSuccess)
  {
      fprintf(stderr,"XOMP acc_init: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);
      exit(err);
  }
  xomp_max_num_devices = rt;  
  assert (rt != -1);
  return xomp_max_num_devices; 
}

/* The default number of devices to be used */
int xomp_get_num_devices (void)
{
  char * env_var_str;
  int  env_var_val;

  //If already initialized, return the value directly
  if (xomp_num_devices!= -1)
  {
//    printf ("DEBUG: xomp_get_num_devices() returns the existing value %d\n", xomp_num_devices);
    return xomp_num_devices;
  }

  // otherwise, obtain it based on max device count and env variable
  if (xomp_max_num_devices == -1)
    xomp_max_num_devices = xomp_get_max_devices();

  env_var_str = getenv("OMP_NUM_DEVICES");
  if (env_var_str != NULL)
  {
    sscanf(env_var_str, "%d", &env_var_val);
    if (env_var_val <= 0) 
    {
       printf ("OMP_NUM_DEVICES should > 0\n");
       exit(1);
    }
    // cap the value
    if (env_var_val > xomp_max_num_devices)
    {
      printf ("OMP_NUM_DEVICES %d is too big, set to max number of devices %d instead\n", env_var_val, xomp_max_num_devices );
      env_var_val = xomp_max_num_devices ;
    }
    xomp_num_devices = env_var_val; 
  }
  else
    xomp_num_devices = xomp_max_num_devices;

//  printf ("DEBUG: xomp_get_num_devices() returns a fresh value %d\n", xomp_num_devices);
  return xomp_num_devices;
}

void omp_set_num_devices (int count)
{
  assert (count>0);
  xomp_num_devices = count; 
}
void xomp_acc_init(void)
{
#if 0  
  hipError_t err;
  int maxDevice = 0;
  err = hipGetDeviceCount(&maxDevice);
  if(err != hipSuccess)
  {
      fprintf(stderr,"XOMP acc_init: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);
      exit(err);
  }
#endif 
  // to be safe, we preallocate memory based on max device count
  xomp_max_num_devices = xomp_get_max_devices();

  DDE_head = (DDE**)calloc(1,sizeof(DDE*)*xomp_max_num_devices);
  DDE_tail = (DDE**)calloc(1,sizeof(DDE*)*xomp_max_num_devices);
  xomp_cuda_prop = (void**)calloc(1,sizeof(void*)*xomp_max_num_devices);
} 

// this can be called multiple times. But the xomp_cuda_prop variable will only be set once
hipDeviceProp_t * xomp_getCudaDeviceProp(int devID)
{
  hipDeviceProp_t* propPointer = NULL;
  if (xomp_cuda_prop[devID] == NULL )
  {
    propPointer = (hipDeviceProp_t *) malloc(sizeof(hipDeviceProp_t));
    xomp_cuda_prop[devID] = propPointer;
    assert (xomp_cuda_prop[devID] != NULL);
    int count;
    hipGetDeviceCount (&count);
    assert (count>=1); // must have at least one GPU here
    
    hipGetDeviceProperties  (propPointer, devID);
  }
  else
    propPointer = (hipDeviceProp_t *)xomp_cuda_prop[devID];
  return propPointer;
}

void xomp_print_gpu_info(int devID)
{
  int max_threads_per_block = xomp_getCudaDeviceProp(devID)->maxThreadsPerBlock;
  int max_blocks_per_grid_x = xomp_getCudaDeviceProp(devID)->maxGridSize[0];
  int global_memory_size =    xomp_getCudaDeviceProp(devID)->totalGlobalMem;
  int shared_memory_size =    xomp_getCudaDeviceProp(devID)->sharedMemPerBlock;
  int registers_per_block =   xomp_getCudaDeviceProp(devID)->regsPerBlock;

  printf ("Found a GPU with \n\tmax threads per block=%d, \n\tmax blocks for Grid X dimension=%d\n\
      \tglobal mem bytes =%d, \n\tshared mem bytes =%d, \n\tregs per block = %d\n",
      max_threads_per_block, max_blocks_per_grid_x, global_memory_size ,  shared_memory_size,
      registers_per_block);
}
// A helper function to probe physical limits based on GPU Compute Capability numbers
// Reference: http://developer.download.nvidia.com/compute/cuda/CUDA_Occupancy_calculator.xls
size_t xomp_get_maxThreadBlocksPerMultiprocessor(int devID)
{
  int major, minor; 
  assert (devID>=0 && devID<xomp_max_num_devices);
  major = xomp_getCudaDeviceProp(devID)-> major;
  minor = xomp_getCudaDeviceProp(devID)-> minor;
  if (major <= 2) //1.x and 2.x: 8 blocks per multiprocessor
    return 8;
  else if (major == 3)
    return 16;
  else if (major == 5)
    return 32;
  else
  {
   printf("Error: xomp_get_maxThreadBlocksPerMultiprocessor(): unhandled Compute Capability numbers%d.%d \n", major, minor);
   assert (false);
  }
  assert (false);
  return 0;
}

// max thread per block, useful for 1-D problem
// The goal is to maximize GPU occupancy for each multiprocessor : physical max warps 
// Reference: http://developer.download.nvidia.com/compute/cuda/CUDA_Occupancy_calculator.xls
//
// Two physical limits are considered for now
//  1) max-active-threads per multiprocessor 
//  2) max active thread blocks per multiprocessor
// So for 1-D block, max threads per block = maxThreadsPerMultiProcessor /  maxBlocks per multiprocessor
size_t xomp_get_maxThreadsPerBlock(int devID)
{
  // this often causes oversubscription to the cores supported by GPU SM processors
  //return xomp_getCudaDeviceProp()->maxThreadsPerBlock;
  //return 128;
  // 2.0: 1536/8= 192 threads per block
  // 3.5 2048/16 = 128
  return xomp_getCudaDeviceProp(devID)->maxThreadsPerMultiProcessor / xomp_get_maxThreadBlocksPerMultiprocessor(devID);
}

/*
* In order to ensure best performance, we setup max_block limitation here, so that each core in the GPU works on only one threads.
* Use XOMP_accelerator_loop_default() runtime to support input data size that exceeds max_block*xomp_get_maxThreadsPerBlock().  
*/
size_t xomp_get_max1DBlock(int devID, size_t s)
{
#if 1  
  size_t block_num = s/xomp_get_maxThreadsPerBlock(devID);
  if (s % xomp_get_maxThreadsPerBlock(devID)!= 0)
     block_num ++;
  //return block_num;     

  size_t max_block = xomp_getCudaDeviceProp(devID)->multiProcessorCount* xomp_get_maxThreadBlocksPerMultiprocessor(devID);

  return block_num<max_block? block_num: max_block; 

  /* max threads per multiprocessor / threads-per-block  * num_multiprocessor */
  //return xomp_getCudaDeviceProp()->multiProcessorCount*(xomp_getCudaDeviceProp()->maxThreadsPerMultiProcessor /xomp_get_maxThreadsPerBlock()) ;
  //return xomp_getCudaDeviceProp()->maxThreadsPerMultiProcessor /xomp_get_maxThreadsPerBlock() ;
#else
  return xomp_getCudaDeviceProp()->multiProcessorCount* xomp_get_maxThreadBlocksPerMultiprocessor();
#endif
}

// Get the max number threads for one dimension (x or y) of a 2D block
// Two factors are considered: the total number of threads within the 2D block must<= total threads per block
//  x * y <= maxThreadsPerBlock 512 or 1024
// each dimension: the number of threads must <= maximum x/y-dimension
//    x <= maxThreadsDim[0],  1024
//    y <= maxThreadsDim[1], 1024 
//  maxThreadsDim[0] happens to be equal to  maxThreadsDim[1] so we use a single function to calculate max segments for both dimensions
size_t xomp_get_max_threads_per_dimesion_2D (int devID)
{

  int max_threads_per_block = xomp_getCudaDeviceProp(devID)->maxThreadsPerBlock;
  // we equalize the number of threads in each dimension
  int max_threads_per_2d_dimension = (int)(sqrt((float)max_threads_per_block));  
  assert (max_threads_per_2d_dimension*max_threads_per_2d_dimension<= max_threads_per_block);

  // our assumption is that dim[0] == dim[1] so we handle x and y in one function
  assert ( xomp_getCudaDeviceProp(devID)->maxThreadsDim[0] == xomp_getCudaDeviceProp(devID)->maxThreadsDim[1]);   
  assert (max_threads_per_2d_dimension <= xomp_getCudaDeviceProp(devID)->maxThreadsDim[0]);
  return max_threads_per_2d_dimension;
}

// return the max number of segments for a dimension (either x or y) of a 2D block
// we define the number of segments to be  SIZE_of_Dimension_x/max_threads_x_dimension
size_t xomp_get_maxSegmentsPerDimensionOf2DBlock(int devID, size_t dimension_size)
{
  // For simplicity, we don't yet consider the factor of warp size for now
  // TODO: block size should be divisible by the warp size??
  // e.g. max threads per block is 1024, then max number of tiles per dimension in a 2D block is 1024^0.5 = 32 threads
  size_t max_threads_per_2d_dimension = xomp_get_max_threads_per_dimesion_2D (devID);
  size_t block_num_x_or_y =  dimension_size/max_threads_per_2d_dimension;
  if (dimension_size % max_threads_per_2d_dimension != 0)
     block_num_x_or_y ++;

  return block_num_x_or_y;
}

/*-----------------------------------------------------
   Device memory allocation functions 
*/
void* xomp_deviceMalloc(size_t size)
{
  void * devPtr;
  hipError_t rt =  hipMalloc(&devPtr, size);
  if ( (size !=0) && (rt == hipSuccess) )
  {
    return devPtr;
  }
  else
  {
    fprintf(stderr, "Error: hipMalloc() failed to allocate the requested %zu bytes!\n", size);
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

// A host version
void* xomp_hostMalloc(size_t size)
{
  assert (size>0);
  void* hostPtr;
  hostPtr = (char*) malloc (size);
  if (hostPtr == NULL)
  {
    fprintf(stderr, "Error: malloc() failed to allocate the requested %zu bytes!\n", size);
    assert (hostPtr != NULL);
  }
  return hostPtr; 
}

// memory copy from src to dest, return the pointer to dest. NULL pointer if anything is wrong 
void * xomp_memcpyHostToDevice (void *dest, const void * src, size_t n)
{
  assert (dest != NULL);
  assert (src != NULL);
  if (xomp_verbose)
    printf("xomp_memcpyHostToDevice(): dest = %p src = %p size = %zu\n", dest, src, n);
  hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyHostToDevice);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy memory from Host %p to Device %p, for %zu bytes!\n", src, dest, n);
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

void * xomp_memcpyDeviceToHost (void *dest, const void * src, size_t n)
{
  assert (dest != NULL);
  assert (src != NULL);
  if (xomp_verbose)
    printf("xomp_memcpyDeviceToHost(): dest = %p src = %p size = %zu\n", dest, src, n);
  hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyDeviceToHost);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy memory from Device %p to Host %p, for %zu bytes!\n", src, dest, n);
    fprintf(stderr, "Error message is =%s\n",hipGetErrorString(rt));
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

// copy a dynamically allocated host source array to a linear dest address on a GPU device. 
// The dimension information of the source array is given by: int dimensions[dimension_size], with known element size. 
// bytes_copied reports the total bytes copied by this function.  
// Liao 4/25/2012
void * xomp_memcpyDynamicHostToDevice (void *dest, const void * src, int * dimensions, size_t dimension_size, size_t element_size, size_t *bytes_copied)
{
  assert (dest != NULL);
  assert (src != NULL);
  if  (dimension_size == 1)  // down to the final, inner-most dimension 
  {
    // this is a firm, bottom count of bytes copied
    *bytes_copied = element_size* dimensions[0]; // number of elements * element_size
    xomp_memcpyHostToDevice (dest, src, *bytes_copied);
  }
  else 
  { 
    assert (dimension_size>=2);
    // 2-D or more: arrays of sub-arrays, copy each sub-array separately
    void ** array2 = (void**) src ; // re-interpret src to be array of arrays
    size_t sub_array_count = dimensions[0];  // top dimension, how many sub-arrays

    // prepare dimension information for sub-arrays
    dimensions ++;// go to the next dimension
    dimension_size --; // sub-array has one less dimension
    char* new_dest = (char*) dest; // byte addressable for dest
    size_t total_subarray_bytes_copied =0; 
    int i; 
    for (i = 0; i< sub_array_count; i++) // for each sub-arrays
    {
      size_t subarray_bytes_copied = 0; 
      void* sub_array_src = (void*) (array2[i]);  // get start address for each sub-array
      assert (sub_array_src != NULL);
      // recursively call to copy each sub-array
      xomp_memcpyDynamicHostToDevice (new_dest, (void*)sub_array_src, dimensions, dimension_size, 
          element_size, &subarray_bytes_copied);

      total_subarray_bytes_copied += subarray_bytes_copied; // recursively accumulate the bytes copied, instead of calculating directly.
      new_dest += subarray_bytes_copied ; // update the dest offset here
    }
    *bytes_copied = total_subarray_bytes_copied; 
  }
  return dest; 
}


// copy linear src memory to dynamically allocated destination, with dimension information given by
// int dimensions[dimension_size]
// the source memory has total n continuous memory, with known size for each element
// the total bytes copied by this function is reported by bytes_copied
// test code: mallocArray-xomp.cu
void * xomp_memcpyDynamicDeviceToHost (void *dest, int * dimensions, size_t dimension_size, const void * src, size_t element_size, size_t *bytes_copied)
{
  // int dimensions[1] = {10}; dimension_size =1; 
  // a[10]: 
  if  (dimension_size == 1)  // down to the final dimension 
  {
    // this is a firm, bottom count of bytes copied
    *bytes_copied = element_size* dimensions[0]; // number of elements * element_size
    xomp_memcpyDeviceToHost (dest, src, *bytes_copied);
  }
  else 
  { 
    int i; 
    assert (dimension_size>=2);
    // 2-D or more: arrays of sub-arrays, copy each sub-array separately
    void ** array2 = (void**) dest; // re-interpret dest to be array of arrays
    size_t sub_array_count = dimensions[0];  // current dimension, how many sub-arrays

    // prepare dimension information for sub-arrays
    dimensions ++;// go to the next dimension
    dimension_size --;
    char* new_src = (char*) src; // byte addressable for src
    size_t total_subarray_bytes_copied =0; 
    for (i = 0; i< sub_array_count; i++) // for each sub-arrays
    {
      size_t subarray_bytes_copied =0; 
      void* sub_array_dest = (void*) (array2[i]);  // get start address for each sub-array

      xomp_memcpyDynamicDeviceToHost ((void*)sub_array_dest,  dimensions, dimension_size, 
          new_src, element_size, &subarray_bytes_copied);

      total_subarray_bytes_copied += subarray_bytes_copied; // recursively accumulate the bytes copied, instead of calculating directly.
      new_src += subarray_bytes_copied ; // update the source offset here
    }
    *bytes_copied = total_subarray_bytes_copied; 
  }
  return dest; 
}

void * xomp_memcpyDeviceToDevice (void *dest, const void * src, size_t n)
{
    hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyDeviceToDevice);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy from Device %p to Device %p for requested %zu bytes!\n", src, dest, n);
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

void * xomp_memcpyHostToHost (void *dest, const void * src, size_t n) // same as memcpy??
{
    hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyHostToHost);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy from Host %p to Host %p for requested %zu bytes!\n", src, dest, n);
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }

}

//------------------------------------------------------
// free the device memory pointed by a pointer, return false in case of failure, otherwise return true
bool xomp_freeDevice(void* devPtr)
{
  hipError_t rt = hipFree (devPtr);
  if (rt == hipSuccess)
    return true;
  else
    return false;
}

// free the host memory pointed by a pointer, return false in case of failure, otherwise return true
bool xomp_freeHost(void* hostPtr)
{
  hipError_t rt = hipHostFree(hostPtr);
  if (rt == hipSuccess)
    return true;
  else
    return false;
}

//------------------------------------------------------
// data set size checking functions
#if 0
  // make sure the length of the array can be mapped to the cuda threads
  assert (SIZE <= max_blocks_per_grid_x* max_threads_per_block);
  // make sure the data will fit into the device memory (shared memory)
  printf("matrix-vector multiplication with size=%d\n", SIZE);
  // one matrix and two vectors
  int mem_required = SIZE*SIZE*sizeof(float) + SIZE* sizeof(float) *2;
  if (global_memory_size > 0) //sometimes the number is too large and it overflows to be a negative integer
    assert (mem_required  <= global_memory_size);
#endif

//------------------------------------------------------
#if 0
double xomp_time_stamp()
{
  struct timeval t;
  double time;
  gettimeofday(&t, NULL);
  time = t.tv_sec + 1.0e-6*t.tv_usec;
  return time;
}
#endif
//------------------------------------------------------
// Host side helper functions
//--- a helper function to allocate 2-D arrays
/* Allocate a multi-dimensional array
 *
 * Input parameters:
 *  int *dimensions:  an integer array storing the size of each dimension
 *  size_t dimension_num: the number of dimensions
 *  size_t esize: the size of an array element
 *
 * return:
 *  the pointer to the allocated array
 * */
void * xomp_mallocArray(int * dimensions, size_t dimension_num, size_t esize)
{
  int i;
  void * array = NULL;

//  if (xomp_verbose)
//    printf("xomp_xomp_mallocArray(): dimensions=%p dimension =%d element size=%d\n",dimensions, dimension_num, esize);
  //  Handle 1-D array: do element-wise malloc
  if ( dimension_num == 1)
  {
    array = (void *) malloc(dimensions[dimension_num - 1]*esize);
    if(array == NULL)
    {
      fprintf(stderr, "out of memory\n");
      abort();
    }
  }
  else // two and more dimensions to be allocated: reduce it to be a 2-step allocation
  {
    // 1st step: allocate the first dimension 
    //           by treating it as allocating a 1-D array of arrays (pointer)
    void ** array2 = NULL;
    array2 = (void **) xomp_mallocArray(dimensions, 1 ,sizeof (void *));
    size_t prev_dim_size = dimensions[0];// number of elements of the first dimension

    // 2nd step: allocate the remaining N -1 dimension arrays, each is an element of the first array 
    // peel off the 1st(previous) dimension, focus on the rest dimensions
     dimensions ++;
    // each of element is an array has a smaller dimension number
     dimension_num --;
    for(i = 0; i < prev_dim_size ; i++)
    {
      array2[i] = xomp_mallocArray (dimensions, dimension_num, esize);
    }
    // return the pointer to the first dimension
    array = (void *) array2;
  }
  return array;
}


/* Free a pointer to a multi-dimensional array
 * int * dimensions: store the sizes of each dimension
 * size_t dimension_num: the number of dimensions
 *
 * */
void xomp_freeArrayPointer (void* array, int * dimensions, size_t dimension_num)
{

  int i;
  // 1-D case, call free() directly
  if (dimension_num == 1)
  {
    free (array);
  }
  else
  { // 2-D or more, iterate through higher dimension and try to free inner arrays
    int prev_dim_size = dimensions [0];
    // step into one dimension
    dimensions ++;
    dimension_num --;
    for (i =0; i< prev_dim_size ; i++)
    {
      xomp_freeArrayPointer (((void **)array)[i], dimensions, dimension_num);
    }
  }
}

#if 0
/*  reduction minus is handled the same way as reduction plus since we just replace the reduction variable with its local copy for each thread
    The associated statement is intact except for the variable replacement : e.g. a-=5 becomes local_a -= 5;
    in the end of each thread accumulates thread local negative values.
    At the block level, we just simply add them all to be the block level negative values
*/
/* we have to encode the type into function name since C function signature does not include parameter list! */
#define XOMP_INNER_BLOCK_REDUCTION_DEF(dtype) \
__device__ void xomp_inner_block_reduction_##dtype(dtype local_value, dtype * grid_level_results, int reduction_op) \
{ \
  /* __shared__ float* sdata[gridDim.x]; not compilable */ \
  /* block size of data, size is specified by the kernel launch parameter (3rd one) */ \
  /* shared data has to have different names for different types. Cannot reuse name across types. */ \
  extern __shared__ dtype sdata_##dtype[];  \
  sdata_##dtype[threadIdx.x] = local_value;  \
  __syncthreads(); \
  /* blockDim.x is the block size */ \
  int isEvenSize = (blockDim.x % 2 ==0); \
  /* contiguous range pattern: half folding and add */ \
  for(int offset = blockDim.x / 2; \
      offset > 0;    /* folding and add */ \
      offset >>= 1) /* offset shrinks half each time */ \
  { \
    if(threadIdx.x < offset)  \
    { \
      /* add a partial sum upstream to our own */ \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
            sdata_##dtype[threadIdx.x] += sdata_##dtype[threadIdx.x + offset]; \
            break; \
         /*  TODO add support for more operations*/ \
         default:  \
            { \
              /* TODO: add assertion or set hipError_t with an error code */ \
              /* cannot call a host function */ \
              /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
              /* assert (false); */ \
             } \
      } /* end switch */ \
    } \
    /* remember to handle the left element */ \
    if ((threadIdx.x == 0) && !isEvenSize) \
    { \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
          sdata_##dtype[0]+= sdata_##dtype[2*offset];  \
          break; \
        /* TODO add more operation support */  \
        default: \
          {  \
            /* TODO: add assertion or set hipError_t with an error code */  \
            /* cannot call a host function */ \
            /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
            /* assert (false); */ \
          } \
      } /* end switch */ \
    } \
    isEvenSize = ( offset % 2 ==0); /* prepare next round*/ \
    /* MUST wait until all threads in the block have updated their partial sums */ \
    __syncthreads(); /* sync after each folding */ \
  } \
  /* thread 0 writes the final result to the partial sum of this thread block */ \
  if(threadIdx.x == 0) \
  { \
    grid_level_results[blockIdx.x] = sdata_##dtype[0]; \
  } \
}

XOMP_INNER_BLOCK_REDUCTION_DEF(int)
XOMP_INNER_BLOCK_REDUCTION_DEF(float)
XOMP_INNER_BLOCK_REDUCTION_DEF(double)

#undef XOMP_INNER_BLOCK_REDUCTION_DEF 

#endif

// TODO: handle more different reduction operations
// TODO : add assertion support
#define XOMP_BEYOND_BLOCK_REDUCTION_DEF(dtype) \
  dtype xomp_beyond_block_reduction_##dtype(dtype * per_block_results, int numBlocks, int reduction_op) \
{ \
  dtype result ; \
  dtype* per_block_results_cpu = (dtype *)xomp_hostMalloc (numBlocks*sizeof(dtype)); \
  xomp_memcpyDeviceToHost (per_block_results_cpu, per_block_results, sizeof(dtype)* numBlocks); \
  int r_i; \
  for (r_i =1; r_i < numBlocks; r_i++) \
  { \
     switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
          per_block_results_cpu[0]+= per_block_results_cpu[r_i]; \
          break; \
        default: \
          { \
          } \
      } \
  } \
  result = per_block_results_cpu[0]; \
  xomp_freeHost(per_block_results_cpu); \
  return result; \
} 

//TODO define more types of CPU level reduction support
XOMP_BEYOND_BLOCK_REDUCTION_DEF(int)
XOMP_BEYOND_BLOCK_REDUCTION_DEF(float)
XOMP_BEYOND_BLOCK_REDUCTION_DEF(double)

#undef XOMP_BEYOND_BLOCK_REDUCTION_DEF 

/* some of the ompacc runtime API */
#if 0
int omp_get_num_devices() {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  return deviceCount;
}
#endif

//! A helper function to copy a mapped variable from src to desc
void copy_mapped_variable (struct XOMP_mapped_variable* desc, struct XOMP_mapped_variable* src)
{
  assert (src != NULL);
  assert (desc != NULL);

  desc-> size = (int*)malloc(sizeof(int) * src->nDim); 
  desc-> offset = (int*)malloc(sizeof(int) * src->nDim); 
  desc-> DimSize = (int*)malloc(sizeof(int) * src->nDim);
  desc->nDim = src->nDim;
  desc->typeSize = src->typeSize;

  desc->address = src->address;
  int i;
  for(i = 0; i < src->nDim; ++i) 
  {
    desc->size[i]= src->size[i]; 
    desc->offset[i]= src->offset[i]; 
    desc->DimSize[i]= src->DimSize[i]; 
  }
  desc->dev_address = src ->dev_address; 
   // we do not want to inherit the copy directions or map-type of parent DDE's variable
   // OpenMP 4.0 has the reuse enclosing data and discard map-type rule.
  //desc->copyFrom= src ->copyFrom; 
}

// create a new DDE-data node and 
// append it to the end of the tracking list, and 
// copy all variables from its parent node to be into the set of inherited variable set.
void xomp_deviceDataEnvironmentEnter(int devID)
{
  // create a new DDE node and initialize it
  DDE * data = (DDE *) malloc (sizeof (DDE));
  assert (data!=NULL);
  data->new_variable_count = 0;
  data->inherited_variable_count = 0;
  data->parent = NULL;
  data->child= NULL;
  data->devID= devID;

  // For simplicity, we pre-allocate the storage for the list of variables
  // TODO: improve the efficiency
  data->new_variables = (struct XOMP_mapped_variable*) malloc (XOMP_MAX_MAPPED_VARS * sizeof (struct XOMP_mapped_variable));
  data->inherited_variables = (struct XOMP_mapped_variable*) malloc (XOMP_MAX_MAPPED_VARS * sizeof (struct XOMP_mapped_variable));

  // Append the data to the list
  // Case 1: empty list, add as the first node, nothing else to do
  if (DDE_tail[devID] == NULL)
  {
    assert (DDE_head[devID] == NULL );
    DDE_head[devID] = data;
    DDE_tail[devID] = data;
    return; 
  }

  // Case 2: non-empty list
  // create double links
  data->parent = DDE_tail[devID]; 
  DDE_tail[devID]->child = data;
  // shift the tail
  DDE_tail[devID] = data;

  // copy all variables from its parent node into the inherited variable set. 
  // Both new and inherited variables of the parent node become inherited for the current node
  data->inherited_variable_count = data->parent->new_variable_count + data->parent->inherited_variable_count;
  data->inherited_variables = (struct XOMP_mapped_variable*) malloc (data->inherited_variable_count * sizeof (struct XOMP_mapped_variable));
  assert (data->inherited_variables != NULL);

  int i;
  int offset = 0;
  for (i = 0; i < data->parent->new_variable_count; i++)
  {
    struct XOMP_mapped_variable* dest_element  = data->inherited_variables + offset;
    DDE* p = data->parent;
    struct XOMP_mapped_variable* src_element  =  p->new_variables + i;

    copy_mapped_variable(dest_element, src_element);
    offset ++;
  }

  for (i = 0; i < data->parent->inherited_variable_count; i++)
  {
    //copy_mapped_variable(&((data->inherited_variables)[offset]), &( (data->parent->inherited_variables)[i]));
    copy_mapped_variable( (struct XOMP_mapped_variable*) (data->inherited_variables + offset), (struct XOMP_mapped_variable*) (data->parent->inherited_variables + i));
    offset ++;
  }
  assert (offset == data->inherited_variable_count);

}

// Check if an original  variable is already mapped in enclosing data environment, return its device variable's address if yes.
// return NULL if not
void* xomp_deviceDataEnvironmentGetInheritedVariable (int devID, void* orig_var, int typeSize, int* size)
{
  void * dev_address = NULL; 
  assert (orig_var != NULL);
  int i; 
  // At this point, DDE list should not be empty
  // At least a call to XOMP_Device_Data_Environment_Enter() should have finished before
  assert ( DDE_tail[devID] != NULL );
  for (i = 0; i < DDE_tail[devID]->inherited_variable_count; i++)
  {
    struct XOMP_mapped_variable* cur_var = DDE_tail[devID]->inherited_variables + i; 
    if (cur_var->address == orig_var)
    {
      dev_address = cur_var-> dev_address;
      int i;
      int matched = 1;
      for(i=0; i < cur_var->nDim; ++i)
      {
        if(cur_var->size[i]*typeSize != size[i]*typeSize)
           matched = 0;
      }
      if(matched)
        break;
    }
  } 
  return dev_address; 
}

//! Add a newly mapped variable into the current DDE's new variable list
void xomp_deviceDataEnvironmentAddVariable (int devID, void* var_addr, int* var_size, int* var_offset, int* var_dim, int nDim, int typeSize, void * dev_addr, bool copyTo, bool copyFrom)
{
  // TODO: sanity check to avoid add duplicated variable or inheritable variable
  assert ( DDE_tail[devID] != NULL );
  struct XOMP_mapped_variable* mapped_var = DDE_tail[devID]->new_variables + DDE_tail[devID]->new_variable_count ;
  mapped_var-> address = var_addr;
  mapped_var-> size = (int*)malloc(sizeof(int) * nDim); 
  mapped_var-> offset = (int*)malloc(sizeof(int) * nDim); 
  mapped_var-> DimSize = (int*)malloc(sizeof(int) * nDim);
  mapped_var->nDim = nDim; 
  mapped_var->typeSize = typeSize; 
  int i;
  for(i = 0; i < nDim; ++i)
  { 
    mapped_var-> size[i] = var_size[i]; 
    mapped_var-> offset[i] = var_offset[i]; 
    mapped_var-> DimSize[i] = var_dim[i]; 
  }
  mapped_var-> dev_address = dev_addr; 
  mapped_var-> copyTo= copyTo; 
  mapped_var-> copyFrom= copyFrom; 
  // now move up the offset
  DDE_tail[devID]->new_variable_count ++;
}

void xomp_memGatherDeviceToHost(void* dest, void* src, int* vsize, int* voffset, int* vDimSize, int ndim, int typeSize)
{
  int offset_src;
  int offset_dest;
  assert (ndim <= 3);
  if(ndim == 1)
  {
     xomp_memcpyDeviceToHost((char*)dest+voffset[0]*typeSize, (char*)src, vsize[0]*typeSize);
  }
  else  if(ndim == 2)
  {
// vsize[1] stores the fastest-access dimension
     int j;
     for(j=0; j < vsize[0]; ++j)
     {
       offset_dest  = voffset[1] + (j + voffset[0]) * vDimSize[1];
       offset_src = j  * vsize[1];
       xomp_memcpyDeviceToHost((char*)dest+offset_dest*typeSize, (char*)src+offset_src*typeSize, vsize[1]*typeSize);
     } 
  }
  else  if(ndim == 3)
  {
     int i,j;
     for(j=0; j < vsize[2]; ++j)
     {
       offset_dest = voffset[0] + vDimSize[0]*( voffset[1] + vDimSize[1] * (j + voffset[2])) - vDimSize[0];
       offset_src = vsize[1] * (j * vsize[2]) - vsize[0];
       for(i=0; i < vsize[1]; ++i)
       {
         offset_dest  += vDimSize[0];
         offset_src += vsize[0];
         xomp_memcpyDeviceToHost((char*)dest+offset_dest*typeSize, (char*)src+offset_src*typeSize, vsize[0]*typeSize);
       } 
     }
  }
}

void xomp_memScatterHostToDevice(void* dest, void* src, int* vsize, int* voffset, int* vDimSize, int ndim, int typeSize)
{
  int offset_src;
  int offset_dest;
  assert (ndim <= 3);
  if(ndim == 1)
  {
     xomp_memcpyHostToDevice((char*)dest, (char*)src+voffset[0]*typeSize, vsize[0]*typeSize);
  }
  else  if(ndim == 2)
  {
     int j;
     for(j=0; j < vsize[0]; ++j)
     {
       offset_src  = voffset[1] + (j + voffset[0]) * vDimSize[1];
       offset_dest = j  * vsize[1];
       xomp_memcpyHostToDevice((char*)dest+offset_dest*typeSize, (char*)src+offset_src*typeSize, vsize[1]*typeSize);
     } 
  }
  else  if(ndim == 3)
  {
     int i,j;
     for(j=0; j < vsize[2]; ++j)
     {
       //offset_src = voffset[0] + vDimSize[0]*( voffset[1] + vDimSize[1] * (j + voffset[2]) -1);
       offset_src = (j+voffset[2])*vDimSize[0]*vDimSize[1] + voffset[1]*vDimSize[0] + voffset[0] - vDimSize[0];
       offset_dest = j * vsize[1] * vsize[2] - vsize[0];
       for(i=0; i < vsize[1]; ++i)
       {
         offset_src  += vDimSize[0];
         offset_dest += vsize[0];
         xomp_memcpyHostToDevice((char*)dest+offset_dest*typeSize, (char*)src+offset_src*typeSize, vsize[0]*typeSize);
       } 
     }
  }
}

// All-in-one function to prepare device variable
void* xomp_deviceDataEnvironmentPrepareVariable(int devID, void* original_variable_address, int nDim, int typeSize, int* vsize, int* voffset, int* vDimSize, bool copy_into, bool copy_back)
{
  // currently only handle one dimension
  void* dev_var_address = NULL; 
  dev_var_address = xomp_deviceDataEnvironmentGetInheritedVariable (devID, original_variable_address, typeSize, vsize);
  if (dev_var_address == NULL)
  {
    int devSize = 1;
    for(int i=0; i < nDim; ++i)
    {
      devSize *= vsize[i];
    }
    dev_var_address = xomp_deviceMalloc(devSize*typeSize);
    xomp_deviceDataEnvironmentAddVariable (devID, original_variable_address, vsize, voffset, vDimSize, nDim, typeSize, dev_var_address, copy_into, copy_back);
    // The spec says : reuse enclosing data and discard map-type rule.
    // So map-type only matters when no-reuse happens
    if (copy_into)
    {
      xomp_memScatterHostToDevice(dev_var_address, original_variable_address, vsize, voffset, vDimSize, nDim, typeSize);
    //  xomp_memcpyHostToDevice(dev_var_address, original_variable_address, vsize[0]);
    }
  }
  assert (dev_var_address != NULL);
  return dev_var_address;
}

// Exit current DDE: copy back values if specified, deallocate memory, delete the DDE-data node from the end of the tracking list
void xomp_deviceDataEnvironmentExit(int devID)
{
  assert ( DDE_tail[devID] != NULL );

  // Deallocate mapped device variables which are allocated by this current DDE
  // Optionally copy the value back to host if specified.
  int i; 
  for (i = 0; i < DDE_tail[devID]->new_variable_count; i++)
  {
    struct XOMP_mapped_variable* mapped_var = DDE_tail[devID]->new_variables + i;
    void * dev_address = mapped_var->dev_address;
    if (mapped_var->copyFrom)
    {
       xomp_memGatherDeviceToHost(((void *)((char*)mapped_var->address)),((void *)((char *)mapped_var->dev_address)), mapped_var->size,mapped_var->offset,mapped_var->DimSize, mapped_var->nDim,mapped_var->typeSize);
       //xomp_memcpyDeviceToHost(((void *)((char*)mapped_var->address+mapped_var->offset[0])),((const void *)mapped_var->dev_address), mapped_var->size[0]);
    }
    // free after copy back!!
    xomp_freeDevice (dev_address); //TODO Will this work without type info? Looks so!
  }

  // Deallocate pre-allocated variable lists
  free (DDE_tail[devID]->new_variables);
  free (DDE_tail[devID]->inherited_variables);
  
  // Delete the node from the tail
  DDE * parent = DDE_tail[devID]->parent; 
  if (parent != NULL)
  {
    assert (DDE_tail[devID] == parent->child); 
    DDE_tail[devID] = parent; 
    free (parent->child);
    parent->child = NULL;
  }
  else // last node in the list
  {
    free (DDE_tail[devID]);
    DDE_head[devID] = NULL;
    DDE_tail[devID] = NULL;
  }  
}


