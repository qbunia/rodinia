#include "hip/hip_runtime.h"
/* 
CUDA and/or nvcc does not have linker for device code.
We have to put some common device functions into this file.
So the generated CUDA file can include the functions as inlined functions.

TODO: extend to support 3-D mapping case, which should be trivial based on 2-D implementation

Modified: 
*Liao, 7/10/2013, extend reduction support for 2-D mapping

Liao 2/11/2013
*/

/*  reduction minus is handled the same way as reduction plus since we just replace the reduction variable with its local copy for each thread
    The associated statement is intact except for the variable replacement : e.g. a-=5 becomes local_a -= 5;
    in the end of each thread accumulates thread local negative values.
    At the block level, we just simply add them all to be the block level negative values
*/
/* we have to encode the type into function name since C function signature does not include parameter list! */
/* grid_level_results: across blocks, each block has a result inside this array of size = gridDim.x * gridDim.y */
#define XOMP_INNER_BLOCK_REDUCTION_DEF(dtype) \
__device__ void xomp_inner_block_reduction_##dtype(dtype local_value, dtype * grid_level_results, int reduction_op) \
{ \
  /* To speedup reduction, we transfer local_value to a shared data within the block */ \
  /* __shared__ float* sdata[blockDim.x * blockDim.y * blockDim.z]; not compilable */ \
  /* block size of data, size is specified by the kernel launch parameter (3rd one) */ \
  /* shared data has to have different names for different types. Cannot reuse name across types. */ \
  extern __shared__ dtype sdata_##dtype[];  \
  /* map 2-D threads into a 1-D shared data: linearization */ \
  int ii = threadIdx.x*blockDim.y + threadIdx.y; \
  sdata_##dtype[ii] = local_value;  \
  __syncthreads(); \
  /* nn: the block size, number of threads per block */ \
  int nn = blockDim.x* blockDim.y ; \
  int isEvenSize = (nn % 2 ==0); \
  /* contiguous range pattern: half folding and add */ \
  for(int offset = nn / 2; \
      offset > 0;    /* folding and add */ \
      offset >>= 1) /* offset shrinks half each time */ \
  { \
    if(ii < offset)  \
    { \
      /* add a partial sum upstream to our own */ \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
            sdata_##dtype[ii] += sdata_##dtype[ii + offset]; \
            break; \
         /*  TODO add support for more operations*/ \
         default:  \
            { \
              /* TODO: add assertion or set hipError_t with an error code */ \
              /* cannot call a host function */ \
              /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
              /* assert (false); */ \
             } \
      } /* end switch */ \
    } \
    /* remember to handle the left element */ \
    if ((ii == 0) && !isEvenSize) \
    { \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
          sdata_##dtype[0]+= sdata_##dtype[2*offset];  \
          break; \
        /* TODO add more operation support */  \
        default: \
          {  \
            /* TODO: add assertion or set hipError_t with an error code */  \
            /* cannot call a host function */ \
            /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
            /* assert (false); */ \
          } \
      } /* end switch */ \
    } \
    isEvenSize = ( offset % 2 ==0); /* prepare next round*/ \
    /* MUST wait until all threads in the block have updated their partial sums */ \
    __syncthreads(); /* sync after each folding */ \
  } \
  /* thread (0,0,0) writes the partial sum of this thread block to grid level results (linearized also)*/ \
  if(ii == 0) \
  { \
    grid_level_results[blockIdx.x*gridDim.y + blockIdx.y] = sdata_##dtype[0]; \
  } \
}

XOMP_INNER_BLOCK_REDUCTION_DEF(int)
XOMP_INNER_BLOCK_REDUCTION_DEF(float)
XOMP_INNER_BLOCK_REDUCTION_DEF(double)

#undef XOMP_INNER_BLOCK_REDUCTION_DEF 

/* Loop scheduling helper functions on GPUs */
//Default loop scheduling, worksharing without any schedule clause
// input upper bound is inclusive (loop normalized with <= or >=)
// output n_upper is also inclusive 
// stride is positive for incremental, negative for decremental iteration space
// Updated on 8/29/2013 Liao
// It turns out that evenly dividing up iteration space for GPU threads is not optimal.
// Each thread may touch a large range (chunk) of data elements , which has low memory access efficiency (cannot be coalesced)
// A better solution is to use round -robin scheduling, similar to the static scheduling with chunk size 1 in regular OpenMP CPU threads loops
__device__ void XOMP_cuda_loop_default_internal(int lower, int upper, int stride, int _p_num_threads, int _p_thread_id, long* n_lower, long* n_upper)
{
  int _p_lower;
  int _p_upper;
  int _p_chunk_size;
  int addOne ; // adjustment to input and output upper bounds, depending on if they are inclusive or non-inclusive

  int isDecremental= 0;
  if (lower>upper)
    isDecremental = 1;

  // calculate loop iteration count from lower, upper and stride , 
  // adjust inclusive stride here TODO
  // no -1/+1? if upper is already an inclusive bound
  int _p_iter_count = 0;
  if (isDecremental == 1)
  {
    addOne = 1;
    //stride = 0 - stride;  // n
    if (stride >0)
    {
/*  TODO: assertion and error code  
      printf("Error: in XOMP_loop_default() of xomp.c: stride must be negative for decremental iteration. stride = %d \n ", stride);
      assert (0);
*/
    }
  }
  else // incremental
  {
    addOne = -1; // real bound should be non-inclusive - 1
    if (stride <0)
    {
/* TODO assertion and error code
      printf("Error: in XOMP_loop_default() of xomp.c: stride must be positive for incremental iteration. stride = %d \n ", stride);
      assert (0);
*/
    }
  }
  // addOne is needed here only if the input upper bound is non-inclusive
  // we use loop normalization to ensure upper bounds are inclusive already.
  // So we don't need addOne here anymore
  //_p_iter_count = ( stride + addOne + upper -  lower) /  stride;
  _p_iter_count = ( stride + upper -  lower) /  stride;

  // calculate a proper chunk size
  // two cases: evenly divisible  20/5 =4
  //   not evenly divisible 20/3= 6
  // Initial candidate  
 // get number of threads of this thread block, 1-D case
//  int _p_num_threads = blockDim.x ;  
  //int _p_num_threads = omp_get_num_threads();

  _p_chunk_size = _p_iter_count / _p_num_threads;
  int _p_ck_temp = _p_chunk_size * _p_num_threads != _p_iter_count;

  _p_chunk_size = _p_ck_temp + _p_chunk_size;

  // decide on the lower and upper bound for the current thread
  //int _p_thread_id = threadIdx.x ;
  //int _p_thread_id = omp_get_thread_num();
//  printf("inside xomp_loop_default(): _p_thread_id =%d\n", _p_thread_id);
  _p_lower =  lower + _p_chunk_size * _p_thread_id *  stride;

  //addOne is needed here if the output upper bound is inclusive
  // -1 if the output n_upper is an inclusive bound, 
  // we do use inclusive in the final normalized loop
  _p_upper = _p_lower + _p_chunk_size *  stride + addOne;

  // adjust inclusive stride here 
  // addOne is needed if the input upper bound is non-inclusive
  // no -1/+1 since upper is already an inclusive bound
  if (isDecremental == 1)
    _p_upper = (_p_upper > (upper ) ?_p_upper : (upper ));
    //_p_upper = (_p_upper > (upper + addOne) ?_p_upper : (upper + addOne));
  else
    _p_upper = (_p_upper < upper ?_p_upper : upper);
    //_p_upper = (_p_upper < (upper + addOne) ?_p_upper : (upper + addOne));

  *n_lower = _p_lower;
  *n_upper = _p_upper;
//  printf("inside xomp_loop_default(): _p_lower=%d, _p_upper=%d\n", _p_lower,_p_upper);
}

/* This is the top level function which decides on the lower and upper bounds for the current thread id and current block id
  It internally calls XOMP_cuda_loop_default_internal twice to figure this out: 
  once for the block's bounds, the other for the thread's bounds
*/
__device__ void XOMP_accelerator_loop_default(int lower, int upper, int stride, long* n_lower, long* n_upper)
{
/* Obtain the lower and upper bounds for the current 1-D thread block*/
 /*  the iteration bounds for this entire thread block */
  long lower_for_block, upper_for_block;
  XOMP_cuda_loop_default_internal (lower, upper, stride, gridDim.x, blockIdx.x, &lower_for_block, &upper_for_block);

 /* now focus on the bounds of the current thread of the current block */
  XOMP_cuda_loop_default_internal (lower_for_block, upper_for_block, stride, blockDim.x, threadIdx.x, n_lower, n_upper);
}

/*
_p_num_threads: number of threads of the thread team participating the scheduling
_p_thread_id: the current thread's id within the current team

  lb and up are inclusive bounds (after normalization)
Return the adjusted numbers including:
  loop_chunk_size: the real chunk size considering original chunksize and step
  loop_sched_index: the lower bound for current thread
  loop_stride: the total stride for one round of scheduling of all threads
*/
__device__ void XOMP_static_sched_init(int lb, int up, int step, int orig_chunk_size, int _p_num_threads, int _p_thread_id, \
              int * loop_chunk_size, int * loop_sched_index, int * loop_stride)
{   
    int nthds = _p_num_threads;

    if (nthds == 1) { // single thread case
      *loop_sched_index = lb;
      //loop_end = up;
      *loop_chunk_size = orig_chunk_size * step;
      *loop_stride = (*loop_chunk_size) * nthds;
      return;
    }

    *loop_chunk_size = orig_chunk_size * step;
    *loop_sched_index = lb + (*loop_chunk_size)* _p_thread_id;
    *loop_stride = (*loop_chunk_size) * nthds;
    //int loop_end = up;
//    int is_last = 0;
}

/*
Using current thread ID (_p_thread_id) and team size (_p_num_threads), calculate lb and ub for the current thread
for the round robin scheduling with lower (loop_sched_index), upper (loop_end) , stride (loop_stride), and chunk size (loop_chunk_size)
*/
__device__ int XOMP_static_sched_next(
    int* loop_sched_index , int loop_end, int orig_step, int loop_stride, int loop_chunk_size,
    int _p_num_threads, int _p_thread_id,
    int *lb,int *ub)
{   
    int b,e;
    b = *loop_sched_index;
  //The code logic is original for exclusive upper bound!!
  // But in ROSE, we normalize all loops to be inclusive bounds. So we have to ajust them in the functions, instead of during transformation.
  //
  // 1. adjust the original loop end from inclusive to be exclusive. 
    if (orig_step >0)
       loop_end ++; // expect the user code will use the upper bound as an inclusive one, so minus one in advance
    else
       loop_end --;

    if (_p_num_threads == 1) { /* not in parallel */
        e = loop_end;
        if(b == e) return 0;
        *lb = b;
        *ub = e;
        *loop_sched_index = e;
#if 1 // need to adjust here!
    if (orig_step >0)
       *ub --; // expect the user code will use the upper bound as an inclusive one, so minus one in advance
    else
       *ub ++;
#endif
        return 1;
    } // thread team has 1 thread only

    *loop_sched_index += loop_stride;

    e = b + loop_chunk_size;
#if 1 // must timely adjust e here !!
    if (orig_step >0)
       e --; // expect the user code will use the upper bound as an inclusive one, so minus one in advance
    else
       e ++;
#endif

    if(loop_chunk_size > 0){
        if(b >= loop_end) return 0;
        if(e >= loop_end){
            e = loop_end;
//            tp->is_last = 1;
        }
    } else {

        if(b <= loop_end) return 0;
#if 0 // too late to adjust, e is already used before!!
        if(e <= tp->loop_end){
            e = tp->loop_end;
            tp->is_last = 1;
        }
#endif
    }
    *lb = b;
    *ub = e;
   return 1;
}

// A wrapper function for  blockDim.x * blockIdx.x + threadIdx.x
// Essentially we just hide CUDA variables (blockDim.x etc) inside this function
// since there are three dimensions x, y, z. we use dimension_no to indicate which dimension is requested.
// dimension_no start from 1 to 3, corresponding to x, y, z dimensions.
__device__ int getLoopIndexFromCUDAVariables(int dimension_no)
{
  if (dimension_no == 1)
   return blockDim.x * blockIdx.x + threadIdx.x;
  else if (dimension_no == 2)
   return blockDim.y * blockIdx.y + threadIdx.y;
  else if (dimension_no == 3)
   return blockDim.z * blockIdx.z + threadIdx.z;
  else
  {
    //printf("getLoopIndexFromCUDAVariables() accept a parameter of range from 1 to 3 only\n");
    //assert (false);
  }
   return -1; 
}

// A wrapper function for gridDim.x * blockDim.x, to hide CUDA variables gridDim.x and blockDim.x.
__device__ int getCUDABlockThreadCount(int dimension_no)
{
   if (dimension_no == 1)
   return gridDim.x * blockDim.x;
  else if (dimension_no == 2)
   return gridDim.y * blockDim.y;
  else if (dimension_no == 3)
   return gridDim.z * blockDim.z;
  else
  {
    //printf("getCUDABlockThreadCount() accept a parameter of range from 1 to 3 only\n");
    //assert (false);
  }
   return -1; 
}


