#include "hip/hip_runtime.h"
#include "axpy.h"

__global__ 
void
axpy_cudakernel_warmingup(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) y[i] += a*x[i];
}

__global__ 
void
axpy_cudakernel_P1(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) y[i] += a*x[i];
}

void axpy_kernel(int N, REAL* Y, REAL* X, REAL a) {
  REAL *d_x, *d_y;
  hipMalloc(&d_x, N*sizeof(REAL));
  hipMalloc(&d_y, N*sizeof(REAL));

  hipMemcpy(d_x, X, N*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_y, Y, N*sizeof(REAL), hipMemcpyHostToDevice);

  // Perform axpy elements
  axpy_cudakernel_warmingup<<<(N+255)/256, 256>>>(d_x, d_y, N, a);
  hipDeviceSynchronize();
  axpy_cudakernel_P1<<<(N+255)/256, 256>>>(d_x, d_y, N, a);
  hipDeviceSynchronize();

  hipMemcpy(Y, d_y, N*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
}

