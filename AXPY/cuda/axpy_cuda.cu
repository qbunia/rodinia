#include "hip/hip_runtime.h"
#include "../axpy.h"

__global__ 
void
axpy_cudakernel_warmingup(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) y[i] += a*x[i];
}

__global__ 
void
axpy_cudakernel_P1(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) y[i] += a*x[i];
}

void axpy_cuda(REAL* x, REAL* y, int n, REAL a) {
  REAL *d_x, *d_y;
  hipMalloc(&d_x, n*sizeof(REAL));
  hipMalloc(&d_y, n*sizeof(REAL));

  hipMemcpy(d_x, x, n*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n*sizeof(REAL), hipMemcpyHostToDevice);

  // Perform axpy elements
  axpy_cudakernel_warmingup<<<(n+255)/256, 256>>>(d_x, d_y, n, a);
  hipDeviceSynchronize();
  axpy_cudakernel_P1<<<(n+255)/256, 256>>>(d_x, d_y, n, a);
  hipDeviceSynchronize();

  hipMemcpy(y, d_y, n*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
}

